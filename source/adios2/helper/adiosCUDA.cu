#include "hip/hip_runtime.h"
/*
 * Distributed under the OSI-approved Apache License, Version 2.0.  See
 * accompanying file Copyright.txt for details.
 *
 * adiosCUDA.cpp
 *
 *  Created on: May 9, 2021
 *      Author: Ana Gainaru gainarua@ornl.gov
 */

#ifndef ADIOS2_HELPER_ADIOSCUDA_CU_
#define ADIOS2_HELPER_ADIOSCUDA_CU_

#include "adios2/common/ADIOSMacros.h"
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>

#include "adiosCUDA.h"

namespace
{
template <class T>
void CUDAMinMaxImpl(const T *values, const size_t size, T &min, T &max)
{
    thrust::device_ptr<const T> dev_ptr(values);
    auto res = thrust::minmax_element(dev_ptr, dev_ptr + size);
    hipMemcpy(&min, thrust::raw_pointer_cast(res.first), sizeof(T),
               hipMemcpyDeviceToHost);
    hipMemcpy(&max, thrust::raw_pointer_cast(res.second), sizeof(T),
               hipMemcpyDeviceToHost);
}
// types non supported on the device
void CUDAMinMaxImpl(const long double * /*values*/, const size_t /*size*/,
                    long double & /*min*/, long double & /*max*/)
{
}
void CUDAMinMaxImpl(const std::complex<float> * /*values*/,
                    const size_t /*size*/, std::complex<float> & /*min*/,
                    std::complex<float> & /*max*/)
{
}
void CUDAMinMaxImpl(const std::complex<double> * /*values*/,
                    const size_t /*size*/, std::complex<double> & /*min*/,
                    std::complex<double> & /*max*/)
{
}
}

template <class T>
void adios2::helper::CUDAMinMax(const T *values, const size_t size, T &min,
                                T &max)
{
    CUDAMinMaxImpl(values, size, min, max);
}

#define declare_type(T)                                                        \
    template void adios2::helper::CUDAMinMax(                                  \
        const T *values, const size_t size, T &min, T &max);
ADIOS2_FOREACH_PRIMITIVE_STDTYPE_1ARG(declare_type)
#undef declare_type

#endif /* ADIOS2_HELPER_ADIOSCUDA_CU_ */
