#include "hip/hip_runtime.h"
/*
 * Distributed under the OSI-approved Apache License, Version 2.0.  See
 * accompanying file Copyright.txt for details.
 *
 * bpStepsWriteReadCuda.cu  Simple example of writing and reading data through ADIOS2 BP engine with
 * multiple simulations steps for every IO step using CUDA
 */

#include <ios>
#include <iostream>
#include <stdexcept> //std::invalid_argument std::exception
#include <string>
#include <vector>

#include <adios2.h>

#include <hip/hip_runtime.h>

__global__ void update_array(float *vect, int val) { vect[blockIdx.x] += val; }

void writer(adios2::ADIOS &adios, const std::string &engine, const std::string &fname,
            const size_t Nx, unsigned int nSteps)
{
    // Initialize the simulation data
    float *gpuSimData;
    hipMalloc(&gpuSimData, Nx * sizeof(float));
    hipMemset(gpuSimData, 0, Nx);

    // Set up the ADIOS structures
    adios2::IO bpIO = adios.DeclareIO("WriteIO");
    bpIO.SetEngine(engine);

    // Declare an array for the ADIOS data of size (NumOfProcesses * Nx)
    const adios2::Dims shape{static_cast<size_t>(Nx)};
    const adios2::Dims start{static_cast<size_t>(0)};
    const adios2::Dims count{Nx};
    auto bpFloats = bpIO.DefineVariable<float>("bpFloats", shape, start, count);
    auto bpStep = bpIO.DefineVariable<unsigned int>("bpStep");

    adios2::Engine bpWriter = bpIO.Open(fname, adios2::Mode::Write);

    // Simulation steps
    for (unsigned int step = 0; step < nSteps; ++step)
    {
        // Make a 1D selection to describe the local dimensions of the
        // variable we write and its offsets in the global spaces
        const adios2::Box<adios2::Dims> sel({0}, {Nx});
        bpFloats.SetSelection(sel);

        // Start IO step every write step
        bpWriter.BeginStep();
        bpFloats.SetMemorySpace(adios2::MemorySpace::GPU);
        bpWriter.Put(bpFloats, gpuSimData);
        bpWriter.Put(bpStep, step);
        bpWriter.EndStep();

        // Update values in the simulation data
        update_array<<<Nx, 1>>>(gpuSimData, 10);
    }

    bpWriter.Close();
}

void reader(adios2::ADIOS &adios, const std::string &engine, const std::string &fname,
            const size_t Nx, unsigned int /*nSteps*/)
{
    // Create ADIOS structures
    adios2::IO bpIO = adios.DeclareIO("ReadIO");
    bpIO.SetEngine(engine);

    adios2::Engine bpReader = bpIO.Open(fname, adios2::Mode::Read);

    unsigned int inStep = 0;
    float *gpuSimData;
    hipMalloc(&gpuSimData, Nx * sizeof(float));
    hipMemset(gpuSimData, 0, Nx);
    for (unsigned int step = 0; bpReader.BeginStep() == adios2::StepStatus::OK; ++step)
    {
        auto bpFloats = bpIO.InquireVariable<float>("bpFloats");
        if (bpFloats)
        {
            const adios2::Dims start{0};
            const adios2::Dims count{Nx};
            const adios2::Box<adios2::Dims> sel(start, count);
            bpFloats.SetSelection(sel);

            bpFloats.SetMemorySpace(adios2::MemorySpace::GPU);
            bpReader.Get(bpFloats, gpuSimData); //, adios2::Mode::Deferred);
        }
        auto bpStep = bpIO.InquireVariable<unsigned int>("bpStep");
        if (bpStep)
        {
            bpReader.Get(bpStep, &inStep);
        }

        bpReader.EndStep();
        if (inStep != step)
        {
            std::cout << "ERROR: step mismatch\n";
            return;
        }
    }
    bpReader.Close();
}

int main(int argc, char **argv)
{
    const int device_id = 1;
    hipSetDevice(device_id);

    const std::string engine = argv[1] ? argv[1] : "BPFile";
    std::cout << "Using engine " << engine << std::endl;

    const std::string filename = engine + "StepsWriteReadCuda.bp";
    const unsigned int nSteps = 10;
    const unsigned int Nx = 6000;
    try
    {
        /** ADIOS class factory of IO class objects */
        adios2::ADIOS adios;

        writer(adios, engine, filename, Nx, nSteps);
        reader(adios, engine, filename, Nx, nSteps);
    }
    catch (std::invalid_argument &e)
    {
        std::cout << "Invalid argument exception, STOPPING PROGRAM\n";
        std::cout << e.what() << "\n";
    }
    catch (std::ios_base::failure &e)
    {
        std::cout << "IO System base failure exception, STOPPING PROGRAM\n";
        std::cout << e.what() << "\n";
    }
    catch (std::exception &e)
    {
        std::cout << "Exception, STOPPING PROGRAM\n";
        std::cout << e.what() << "\n";
    }

    return 0;
}
