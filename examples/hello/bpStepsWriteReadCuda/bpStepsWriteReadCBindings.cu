#include "hip/hip_runtime.h"
/*
 * Distributed under the OSI-approved Apache License, Version 2.0.  See
 * accompanying file Copyright.txt for details.
 *
 * bpStepsWriteReadCBindings.cu  Simple example of writing and reading data through ADIOS2 BP engine
 * with multiple simulations steps for every IO step using CUDA (using the C bindings)
 */

#include <adios2_c.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void update_array(float *vect, int val) { vect[blockIdx.x] += val; }

void writer(adios2_adios *adios, const char *fname, const size_t Nx, unsigned int nSteps)
{
    // Initialize the simulation data
    float *gpuSimData;
    hipMalloc(&gpuSimData, Nx * sizeof(float));
    hipMemset(gpuSimData, 0, Nx);

    // Set up the ADIOS structures
    adios2_io *bpIO = adios2_declare_io(adios, "WriteIO");

    size_t shape[1];
    size_t start[1];
    size_t count[1];
    shape[0] = Nx;
    start[0] = 0;
    count[0] = Nx;

    adios2_variable *bpFloats = adios2_define_variable(
        bpIO, "bpFloats", adios2_type_float, 1, shape, start, count, adios2_constant_dims_true);

    adios2_engine *bpWriter = adios2_open(bpIO, fname, adios2_mode_write);

    adios2_step_status err;
    for (unsigned int step = 0; step < nSteps; ++step)
    {
        adios2_begin_step(bpWriter, adios2_step_mode_append, 0.0f, &err);
        adios2_set_memory_space(bpFloats, adios2_memory_space_gpu);
        adios2_put(bpWriter, bpFloats, gpuSimData, adios2_mode_sync);
        adios2_end_step(bpWriter);

        // Update values in the simulation data
        update_array<<<Nx, 1>>>(gpuSimData, 10);
    }

    adios2_close(bpWriter);
    hipFree(gpuSimData);
}

void reader(adios2_adios *adios, const char *fname, const size_t Nx, unsigned int nSteps)
{
    adios2_step_status status;

    adios2_io *bpIO = adios2_declare_io(adios, "ReadIO");

    adios2_engine *bpReader = adios2_open(bpIO, fname, adios2_mode_read);

    float *gpuSimData;
    hipMalloc(&gpuSimData, Nx * sizeof(float));
    hipMemset(gpuSimData, 0, Nx);

    while (adios2_begin_step(bpReader, adios2_step_mode_read, -1., &status) == adios2_error_none)
    {
        if (status == adios2_step_status_end_of_stream)
        {
            break;
        }

        adios2_variable *bpFloats = adios2_inquire_variable(bpIO, "bpFloats");
        size_t start[1];
        size_t count[1];
        start[0] = 0;
        count[0] = Nx;
        adios2_set_selection(bpFloats, 1, start, count);
        adios2_set_memory_space(bpFloats, adios2_memory_space_gpu);
        adios2_get(bpReader, bpFloats, gpuSimData, adios2_mode_sync);
        adios2_end_step(bpReader);
    }
    adios2_close(bpReader);
    hipFree(gpuSimData);
}

int main(int argc, char **argv)
{
    const int device_id = 1;
    hipSetDevice(device_id);

    const char filename[30] = "BPStepsWriteReadCBindings.bp";
    const unsigned int nSteps = 2;
    const unsigned int Nx = 3;

    adios2_adios *adios = adios2_init_serial();
    writer(adios, filename, Nx, nSteps);
    reader(adios, filename, Nx, nSteps);

    return 0;
}
