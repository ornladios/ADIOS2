#include "hip/hip_runtime.h"
/*
 * Simple example of writing and reading data
 * through ADIOS2 BP engine with multiple simulations steps
 * for every IO step.
 */

#include <ios>
#include <iostream>
#include <vector>

#include <adios2.h>

#include <hip/hip_runtime.h>

__global__ void update_array(float *vect, int val) { vect[blockIdx.x] += val; }

int BPWrite(const std::string fname, const size_t N, int nSteps)
{
    // Initialize the simulation data
    float *gpuSimData;
    hipMalloc(&gpuSimData, N * sizeof(float));
    hipMemset(gpuSimData, 0, N);

    // Set up the ADIOS structures
    adios2::ADIOS adios;
    adios2::IO io = adios.DeclareIO("WriteIO");
    io.SetEngine("BPFile");

    // Declare an array for the ADIOS data of size (NumOfProcesses * N)
    const adios2::Dims shape{static_cast<size_t>(N)};
    const adios2::Dims start{static_cast<size_t>(0)};
    const adios2::Dims count{N};
    auto data = io.DefineVariable<float>("data", shape, start, count);

    adios2::Engine bpWriter = io.Open(fname, adios2::Mode::Write);

    // Simulation steps
    for (size_t step = 0; step < nSteps; ++step)
    {
        // Make a 1D selection to describe the local dimensions of the
        // variable we write and its offsets in the global spaces
        adios2::Box<adios2::Dims> sel({0}, {N});
        data.SetSelection(sel);

        // Start IO step every write step
        bpWriter.BeginStep();
        data.SetMemorySpace(adios2::MemorySpace::GPU);
        bpWriter.Put(data, gpuSimData);
        bpWriter.EndStep();

        // Update values in the simulation data
        update_array<<<N, 1>>>(gpuSimData, 10);
    }

    bpWriter.Close();
    return 0;
}

int BPRead(const std::string fname, const size_t N, int nSteps)
{
    // Create ADIOS structures
    adios2::ADIOS adios;
    adios2::IO io = adios.DeclareIO("ReadIO");
    io.SetEngine("BPFile");

    adios2::Engine bpReader = io.Open(fname, adios2::Mode::Read);
    auto data = io.InquireVariable<float>("data");
    std::cout << "Steps expected by the reader: " << bpReader.Steps()
              << std::endl;
    std::cout << "Expecting data per step: " << data.Shape()[0];
    std::cout << " elements" << std::endl;

    int write_step = bpReader.Steps();
    // Create the local buffer and initialize the access point in the ADIOS file
    std::vector<float> simData(N); // set size to N
    const adios2::Dims start{0};
    const adios2::Dims count{N};
    const adios2::Box<adios2::Dims> sel(start, count);
    data.SetSelection(sel);

    float *gpuSimData;
    hipMalloc(&gpuSimData, N * sizeof(float));
    hipMemset(gpuSimData, 0, N);
    // Read the data in each of the ADIOS steps
    for (size_t step = 0; step < write_step; step++)
    {
        data.SetStepSelection({step, 1});
        data.SetMemorySpace(adios2::MemorySpace::GPU);
        bpReader.Get(data, gpuSimData, adios2::Mode::Deferred);
        bpReader.PerformGets();
        hipMemcpy(simData.data(), gpuSimData, N * sizeof(float),
                   hipMemcpyDeviceToHost);
        std::cout << "Simualation step " << step << " : ";
        std::cout << simData.size() << " elements: " << simData[1] << std::endl;
    }

    bpReader.Close();
    return 0;
}

int main(int argc, char **argv)
{
    const std::string fname("CudaBp4wr.bp");
    const int device_id = 1;
    hipSetDevice(device_id);
    const size_t N = 6000;
    int nSteps = 10, ret = 0;

    ret += BPWrite(fname, N, nSteps);
    ret += BPRead(fname, N, nSteps);
    return ret;
}
